
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <utility>
#include <ctime>
#include <time.h>

using namespace std;
// output
ofstream fo("Ans.txt");

// Cac bien hang so
const int ARRAY_SIZE_INP = 12005;
const int ARRAY_BYTES_INP = ARRAY_SIZE_INP * sizeof(int);
const int ARRAY_SIZE_OUT = 605;
const int ARRAY_BYTES_OUT = ARRAY_SIZE_OUT * sizeof(int);

//cac bien chinh
int l = 9, d = 2;
char cDataInp[ARRAY_SIZE_INP];
int h_dataMotif[ARRAY_SIZE_INP];
string sDataInp[20];

struct Motif_Ans
{
	int dis;
	string motif;
	int adress[20];
};

//input tu file
void File_Input()
{
	l = 9; d = 2;
	FILE * pFile;
	pFile = fopen("datacu.txt", "r");
	if (pFile == NULL)
		perror("Error opening file");
	else
	{
		if (fgets(cDataInp, ARRAY_SIZE_INP, pFile) != NULL)
			cout << "nhap du lieu thanh cong!\n";
		fclose(pFile);
	}

	for (int i = 0; i < strlen(cDataInp); ++i) {
		//A=0 C=1 G=2 T=3
		switch (cDataInp[i])
		{
		case 'A': { h_dataMotif[i] = 0; break; }
		case 'C': { h_dataMotif[i] = 1; break; }
		case 'G': { h_dataMotif[i] = 2; break; }
		case 'T': { h_dataMotif[i] = 3; break; }
		default: cout << "error chuyen sang int";
			break;
		}
	}
	int k = 0;
	string temp = cDataInp;
	//cout << temp << endl;
	for (int i = 0; i < temp.size(); i += 600) {
		sDataInp[k] = temp.substr(i, 600);
		//cout << k << ". " << sDataInp[k] << endl;
		k++;
	}
}

int score_ham(string s1, string s2)
{
	int res = 0;
	for (int i = 0; i<s1.size(); ++i) if (s1[i] != s2[i]) res++;
	return res;
}

Motif_Ans dis_hamming(string s)
{
	Motif_Ans res;
	res.motif = s;
	int res_Sum = 0, temp_score = 999, temp_Adress;
	for (int i = 0; i<20; ++i)
	{
		string s1 = sDataInp[i];
		temp_score = 999;
		for (int j = 0; j < s1.size() - l + 1; ++j)
		{
			string temp_str = s1.substr(j, l);
			int score_s = score_ham(s, temp_str);
			if (score_s < temp_score)
			{
				temp_score = score_s;
				temp_Adress = j + 1;
			}
		}
		res_Sum += temp_score;
		res.adress[i] = temp_Adress;
	}
	res.dis = res_Sum;
	return res;
}

__global__ void patternBarching(const int* d_datainp, const int l, const int d, int *ans) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < 600 - l) {
		//printf("\n %d", index);
		int ansMotif_sorce = 999;// motif tra ra
		int ansMotif_string[40];//motif tra ra

		int motif_NeSorce = 999;//kq tra ve ham NE
		int motif_NeString[40];//kq tra ve ham NE
		int temp_Sorce = 999;
		int temp_Str[40];

		//cat chuoi motif
		for (int i = 0; i < l; ++i) {
			ansMotif_string[i] = d_datainp[i + index];
			motif_NeString[i] = ansMotif_string[i];
		}
		//begin tinh hamming
		int tempRow, tempSubRow;
		for (int i = 0; i < 20; ++i)
		{
			tempRow = 999;
			for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
			{
				tempSubRow = 0;
				for (int k = 0; k < l; k++) {
					if (ansMotif_string[k] != d_datainp[k + j]) tempSubRow++;
				}
				if (tempSubRow < tempRow) tempRow = tempSubRow;
			}
			ansMotif_sorce += tempRow;
		}
		//end tinh hamming cho chuoi vao

		//begin tinh pattern branching
		for (int a = 0; a <= d; a++) {
			//kiem tra motif dis
			if (motif_NeSorce < ansMotif_sorce) {
				ansMotif_sorce = motif_NeSorce;
				for (int i = 0; i < l; ++i) {
					ansMotif_string[i] = motif_NeString[i];
					temp_Str[i] = motif_NeString[i];
				}
			}
			else
			{//gan bien Ham Ne
				for (int i = 0; i < l; ++i) {
					temp_Str[i] = ansMotif_string[i];
				}
			}//end kiem tra motif

			//begin ham bestNeighbor
			int change = -1;
			for (int b = 0; b < l; ++b) {
				//trg hop 0 A
				if (temp_Str[b] != 0) {
					change = temp_Str[b];
					temp_Str[b] = 0;
					temp_Sorce = 0;//diem dis
					//begin tinh hamming
					for (int i = 0; i < 20; ++i)
					{
						tempRow = 999;
						for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
						{
							tempSubRow = 0;
							for (int k = 0; k < l; k++) {
								if (temp_Str[k] != d_datainp[k + j]) tempSubRow++;
							}
							if (tempSubRow < tempRow) tempRow = tempSubRow;
						}
						temp_Sorce += tempRow;
					}
					//end tinh hamming cho chuoi vao
					//kiem tra dis motif Ne
					if (temp_Sorce < motif_NeSorce) {
						motif_NeSorce = temp_Sorce;
						for (int c = 0; c < l; ++c) {
							motif_NeString[c] = temp_Str[c];
						}
					}
					temp_Str[b] = change;//tra lai gia tri ban dau
				}
				//trg hop 1 C
				if (temp_Str[b] != 1) {
					change = temp_Str[b];
					temp_Str[b] = 1;
					temp_Sorce = 0;//diem dis
					//begin tinh hamming
					for (int i = 0; i < 20; ++i)
					{
						tempRow = 999;
						for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
						{
							tempSubRow = 0;
							for (int k = 0; k < l; k++) {
								if (temp_Str[k] != d_datainp[k + j]) tempSubRow++;
							}
							if (tempSubRow < tempRow) tempRow = tempSubRow;
						}
						temp_Sorce += tempRow;
					}
					//end tinh hamming cho chuoi vao
					//kiem tra dis motif Ne
					if (temp_Sorce < motif_NeSorce) {
						motif_NeSorce = temp_Sorce;
						for (int c = 0; c < l; ++c) {
							motif_NeString[c] = temp_Str[c];
						}
					}
					temp_Str[b] = change;
				}
				//trg hop 2 G
				if (temp_Str[b] != 2) {
					change = temp_Str[b];
					temp_Str[b] = 2;
					temp_Sorce = 0;//diem dis
								   //begin tinh hamming
					for (int i = 0; i < 20; ++i)
					{
						tempRow = 999;
						for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
						{
							tempSubRow = 0;
							for (int k = 0; k < l; k++) {
								if (temp_Str[k] != d_datainp[k + j]) tempSubRow++;
							}
							if (tempSubRow < tempRow) tempRow = tempSubRow;
						}
						temp_Sorce += tempRow;
					}
					//end tinh hamming cho chuoi vao
					//kiem tra dis motif Ne
					if (temp_Sorce < motif_NeSorce) {
						motif_NeSorce = temp_Sorce;
						for (int c = 0; c < l; ++c) {
							motif_NeString[c] = temp_Str[c];
						}
					}
					temp_Str[b] = change;
				}
				//trg hop 3 T
				if (temp_Str[b] != 3) {
					change = temp_Str[b];
					temp_Str[b] = 3;
					temp_Sorce = 0;//diem dis
								   //begin tinh hamming
					for (int i = 0; i < 20; ++i)
					{
						tempRow = 999;
						for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
						{
							tempSubRow = 0;
							for (int k = 0; k < l; k++) {
								if (temp_Str[k] != d_datainp[k + j]) tempSubRow++;
							}
							if (tempSubRow < tempRow) tempRow = tempSubRow;
						}
						temp_Sorce += tempRow;
					}
					//end tinh hamming cho chuoi vao
					//kiem tra dis motif Ne
					if (temp_Sorce < motif_NeSorce) {
						motif_NeSorce = temp_Sorce;
						for (int c = 0; c < l; ++c) {
							motif_NeString[c] = temp_Str[c];
						}
					}
					temp_Str[b] = change;
				}
			}
		}//end Ne
		//end tinh

		int dem = 0;
		int res = 0;
		for (int i = 0; i < l; ++i) {
			res = res | (ansMotif_string[i] << dem);
			dem += 2;
			if (index == 574) printf("%d ", ansMotif_string[i]);
		}
		ans[index] = res;
	}
}


int main()
{
	File_Input();

	//test
	/*string test = "GTTCGGCGT";
	Motif_Ans testMoitf = dis_hamming(test);
	fo << testMoitf.dis << endl;
	cout<<sDataInp[0].substr(574, l) << endl;
	cout << h_dataMotif[574] << endl;*/
	//end test
	int h_dataOut[ARRAY_SIZE_OUT];
	for (int i = 0; i < 600; ++i) {
		h_dataOut[i] = -1;
	}
	//GPU khoi tao bien va bo nho
	int *d_dataMotif;
	if (hipMalloc(&d_dataMotif, ARRAY_BYTES_INP) != hipSuccess) {
		cout << "error allocating memory!" << endl;
		return 0;
	}
	int *d_dataOut;
	if (hipMalloc(&d_dataOut, ARRAY_BYTES_OUT) != hipSuccess) {
		cout << "error allocating memory!" << endl;
		hipFree(d_dataMotif);
		return 0;
	}
	if (hipMemcpy(d_dataMotif, h_dataMotif, ARRAY_BYTES_INP, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "error copying memory!" << endl;
		hipFree(d_dataMotif);
		hipFree(d_dataOut);
		return 0;
	}
	if (hipMemcpy(d_dataOut, h_dataOut, ARRAY_BYTES_OUT, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "error copying memory!" << endl;
		hipFree(d_dataMotif);
		hipFree(d_dataOut);
		return 0;
	}

	cout << "dang chay ...." << endl;

	//khoi tao chay cuda
	int threadsPerBlock = 256;
	int blocksPerGrid = (600 + threadsPerBlock - 1) / threadsPerBlock;
	patternBarching <<<blocksPerGrid, threadsPerBlock >>> (d_dataMotif, l, d, d_dataOut);

	fo << "\nTime " << clock() / (double)1000 << " Sec" << endl;

	//copy data tro ve
	if (hipMemcpy(h_dataOut, d_dataOut, ARRAY_BYTES_OUT, hipMemcpyDeviceToHost) != hipSuccess) {
		cout << "error copying memory!" << endl;
		hipFree(d_dataMotif);
		hipFree(d_dataOut);
		return 0;
	}
	//lay best motif
	cout << "\n du lieu tra ve" << endl;
	Motif_Ans best_motif,temp_motif_return;
	best_motif.dis = 999;
	for (int i = 0; i < 600; i++)
	{
		int chuyenStr = h_dataOut[i];
		int k = 0;
		string res = "";
		//cout << chuyenStr << endl;
		if (chuyenStr != -1) {
			//chuyen kieu in sang string
			for (int j = 0; j < l; ++j) {
				int temp = (chuyenStr >> k) & 3;
				//cout << temp << ' ';
				switch (temp)
				{
				case 0:
				{
					res += 'A'; break;
				}
				case 1:
				{
					res += 'C'; break;
				}
				case 2:
				{
					res += 'G'; break;
				}
				case 3:
				{
					res += 'T'; break;
				}
				}
				k += 2;
			}
			if (i == 574) fo << res << endl;
			//ket thuc chuyen
			//kiem tra do dai va tra vi tri
			temp_motif_return = dis_hamming(res);
			if (temp_motif_return.dis < best_motif.dis) {
				cout << "thay doi best" << endl;
				best_motif.dis = temp_motif_return.dis;
				best_motif.motif = temp_motif_return.motif;
				for (int z = 0; z < 20; ++z) {
					best_motif.adress[z] = temp_motif_return.adress[z];
				}
			}
			//end kiem tra
			cout << "------------" << endl;
			cout << temp_motif_return.motif << endl;
			cout << temp_motif_return.dis << endl;
			cout << best_motif.motif << endl;
			cout << best_motif.dis << endl;
			cout << "+++++++++++++" << endl;
		}
	}
	fo << "Best motif: " << best_motif.motif << endl << "Motif location: " << endl;
	for (int z = 0; z < 20; ++z) {
		fo << best_motif.adress[z] << ' ';
	}
	cout << "xong" << endl;

	hipFree(d_dataMotif);
	hipFree(d_dataOut);
	return 0;
}