
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <utility>

using namespace std;
// output
ofstream fo("Ans.txt");

// Cac bien hang so
const int ARRAY_SIZE = 12005;
const int ARRAY_BYTES_INT = ARRAY_SIZE * sizeof(int);

//cac bien chinh
int l = 9, d = 3;
char cDataInp[ARRAY_SIZE];
int h_dataMotif[12005];
string sDataInp[20];

//input tu file
void File_Input()
{
	l = 9; d = 2;
	FILE * pFile;
	pFile = fopen("datacu.txt", "r");
	if (pFile == NULL)
		perror("Error opening file");
	else
	{
		if (fgets(cDataInp, ARRAY_SIZE, pFile) != NULL)
			cout << "nhap du lieu thanh cong!\n";
		fclose(pFile);
	}

	for (int i = 0; i < strlen(cDataInp); ++i) {
		//A=0 C=1 G=2 T=3
		switch (cDataInp[i])
		{
		case 'A': { h_dataMotif[i] = 0; break; }
		case 'C': { h_dataMotif[i] = 1; break; }
		case 'G': { h_dataMotif[i] = 2; break; }
		case 'T': { h_dataMotif[i] = 3; break; }
		default: cout << "error chuyen sang int";
			break;
		}
	}
	int k = 0;
	string temp = cDataInp;
	cout << temp << endl;
	for (int i = 0; i < temp.size(); i += 600) {
		sDataInp[k] = temp.substr(i, 600);
		cout << k << ". " << sDataInp[k] << endl;
		k++;
	}
}

__global__ void patternBarching(const int* d_datainp, const int l, const int d, int *ans) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < 600 - l) {
		//khai bao bien
		int motif_temp[40];
		int temp_val;
		int temp_dis;
		int best_dis = 99999;
		int motif_bN[40];
		int score_motif;
		int s1[40];

		//lay chuoi can duyet
		for (int i = 0; i < l; ++i) {
			motif_temp[i] = d_datainp[i + index];
			motif_bN[i] = motif_temp[i];
			s1[i] = motif_temp[i];
		}
		//ham dis_hamming
		int ans_Ham = 0;
		int temp, tempRow;
		for (int i = 0; i < 20; ++i)
		{
			tempRow = 999;
			for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
			{
				temp = 0;
				for (int k = 0; k < l; k++) {
					if (s1[k] != d_datainp[k + j]) temp++;
				}
				if (temp < tempRow) tempRow = temp;
			}
			ans_Ham += tempRow;
		}
		int sorce_Hamming = ans_Ham;
		//End ham hamming

		//chay ham patternBarching
		for (int k = 0; k < d; ++k) {
			//kiem tra chuoi tot
			//printf("\n 2 \n");
			if (best_dis < score_motif) {
				score_motif = best_dis;
				for (int i = 0; i < l; ++i) {
					motif_temp[i] = motif_bN[i];
				}
			}
			//ham bestNeighbor
			//printf("\nbestNeighbor\n");
			for (int i = 0; i < l; ++i) {
				//printf("\n 3 \n");
				//trg hop 0
				if (motif_temp[i] != 0) {
					temp_val = motif_temp[i];
					motif_temp[i] = 0;
					//temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 1
				if (motif_temp[i] != 1) {
					temp_val = motif_temp[i];
					motif_temp[i] = 1;
					//temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 2
				if (motif_temp[i] != 2) {
					temp_val = motif_temp[i];
					motif_temp[i] = 2;
					//temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 3
				if (motif_temp[i] != 3) {
					temp_val = motif_temp[i];
					motif_temp[i] = 3;
					//temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
			}
			// END ham bestNeighbor
		}
		//printf("\n 4 \n");
		//du lieu tra lai
		//printf("\n gan du lieu vao d_motif \n");
		//End ham
	}
}

int main()
{
	File_Input();
	return 0;
}